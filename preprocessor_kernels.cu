#include "hip/hip_runtime.h"
#include "preprocessor_kernels.h"
#include "preprocessor.h"
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/inner_product.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include "histogram.cu"


//* 1st version of kernel
__global__ void assign_scores_kernel ( int * d_authorized_caldidates_array, 
   int * d_histogram_array, long * d_scores_array, int num_vars ) {
   
   int tid = threadIdx.x + blockDim.x*blockIdx.x; 
   int stride = gridDim.x*blockDim.x;

   while ( tid < num_vars ) {
      
      int x = tid+1;
      d_authorized_caldidates_array[x] = x;
      int h_x_p = d_histogram_array[2*x];
      int h_x_n = d_histogram_array[2*x-1];
      if (  h_x_p == 0 || h_x_n == 0 ) {
         d_scores_array[x] = max(h_x_p, h_x_n);
      } else {
         d_scores_array[x] = h_x_p * h_x_n;
      }
      tid = tid + stride;

   }

}


extern "C" void run_assign_scores_kernel ( int * authorized_caldidates_array, 
   int * histogram_array, int * scores_array, int num_vars ) {
   
   int * d_authorized_caldidates_array;
   int * d_histogram_array;
   long * d_scores_array;
   
   int size_of_authorized_caldidates_array = sizeof(int)*(num_vars+1);
   int size_of_histogram_array = sizeof(int)*(2*num_vars+1);
   int size_of_scores_array = sizeof(long)*(num_vars+1);
   
   hipMalloc ( ( void ** ) &d_authorized_caldidates_array, 
      size_of_authorized_caldidates_array );
   hipMalloc ( ( void ** ) &d_histogram_array, 
      size_of_histogram_array );
   hipMalloc ( ( void ** ) &d_scores_array, 
      size_of_scores_array );
   
   hipMemcpy ( (void*) d_histogram_array, 
      (void*) histogram_array, size_of_histogram_array, 
      hipMemcpyHostToDevice );

   //* TODO: fix num_blocks, it is too huge right now (millions of vars are possible)
   int num_blocks = ceil( ((double)num_vars) / ((double)256));
   int block_size = 256;
   

   hipEvent_t start_gpu_1, end_gpu_1;
   float msecs_gpu_1;
   hipEventCreate(&start_gpu_1);
   hipEventCreate(&end_gpu_1);
   hipEventRecord(start_gpu_1, 0);
      assign_scores_kernel <<< num_blocks, block_size >>> ( d_authorized_caldidates_array, 
         d_histogram_array, d_scores_array, num_vars );
      hipDeviceSynchronize();
   hipEventRecord(end_gpu_1, 0);
   hipEventSynchronize(end_gpu_1);
   hipEventElapsedTime(&msecs_gpu_1, start_gpu_1, end_gpu_1);
   hipEventDestroy(start_gpu_1);
   hipEventDestroy(end_gpu_1);
   std::cout<<"\nassign_scores_kernel done in "<<msecs_gpu_1<<" milliseconds.\n";


   hipMemcpy ( (void*) authorized_caldidates_array, 
      (void*) d_authorized_caldidates_array, 
      size_of_authorized_caldidates_array, hipMemcpyDeviceToHost );
   hipMemcpy ( (void*) scores_array, 
      (void*) d_scores_array, 
      size_of_scores_array, hipMemcpyDeviceToHost );

}


void Preprocessor::run_create_histogram_array_kernel() {
  
   hipEvent_t start_gpu_1, end_gpu_1;
   float msecs_gpu_1;
   hipEventCreate(&start_gpu_1);
   hipEventCreate(&end_gpu_1);
   hipEventRecord(start_gpu_1, 0);
         thrust::device_vector<int> final_histogram;
         final_histogram.resize(2*num_vars+1);      
         for(int i=0; i<cnf->getNumClauses(); i++) {
            Clause c = cnf->getClause(i);
            thrust::device_vector<int> input_array( c.getClauseAsArray(), c.getClauseAsArray() + c.getNumLits());
            thrust::device_vector<int> histogram;
            histogram.resize(2*num_vars+1);
            dense_histogram(input_array, histogram);
            thrust::transform(histogram.begin(), histogram.end(), final_histogram.begin(), final_histogram.begin(), thrust::plus<int>());
         }
         thrust::copy(final_histogram.begin(), final_histogram.end(), histogram_array);
   hipEventRecord(end_gpu_1, 0);
   hipEventSynchronize(end_gpu_1);
   hipEventElapsedTime(&msecs_gpu_1, start_gpu_1, end_gpu_1);
   hipEventDestroy(start_gpu_1);
   hipEventDestroy(end_gpu_1);
   std::cout<<"\nrun_create_histogram_array_kernel done in "<<msecs_gpu_1<<" milliseconds.\n";

}

void Preprocessor::run_sort_wrt_scores_kernel(){
   hipEvent_t start_gpu_1, end_gpu_1;
   float msecs_gpu_1;
   hipEventCreate(&start_gpu_1);
   hipEventCreate(&end_gpu_1);
   hipEventRecord(start_gpu_1, 0);
         thrust::sort_by_key(scores_array + 1, scores_array + num_vars + 1, authorized_caldidates_array + 1);
   hipEventRecord(end_gpu_1, 0);
   hipEventSynchronize(end_gpu_1);
   hipEventElapsedTime(&msecs_gpu_1, start_gpu_1, end_gpu_1);
   hipEventDestroy(start_gpu_1);
   hipEventDestroy(end_gpu_1);
   std::cout<<"\nrun_sort_wrt_scores_kernel done in "<<msecs_gpu_1<<" milliseconds.\n";
}